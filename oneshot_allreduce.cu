#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// #include "src/fastertransformer/utils/cuda_type_utils.cuh"
#include "hip/hip_fp16.h"
#include "assert.h"
#include "vector"
#include "memory"

#define CUSTOM_AR_SIZE_THRESHOLD 50331648
#define MAX_ALL_REDUCE_BLOCKS 24
#define FLAG(a) ((uint32_t)((a) % 0x146))
#define RANKS_PER_NODE 8
#define WARP_SIZE 32
#define DEFAULT_BLOCK_SIZE 1024
#define DEFALUT_ALGO_AR_SIZE_THRESHOLD 196608


static const char* _cudaGetErrorEnum(hipError_t error)
{
    return hipGetErrorString(error);
}

template<typename T>
void check(T result, char const* const func, const char* const file, int const line)
{
    if (result) {
        throw std::runtime_error(std::string("[FT][ERROR] CUDA runtime error: ") + (_cudaGetErrorEnum(result)) + " "
                                 + file + ":" + std::to_string(line) + " \n");
    }
}

#define check_cuda_error(val) check((val), #val, __FILE__, __LINE__)


#ifdef ENABLE_BF16
typedef struct bf168 {
    __hip_bfloat162 x;
    __hip_bfloat162 y;
    __hip_bfloat162 z;
    __hip_bfloat162 w;
} bf168;
#endif

template<typename T>
struct AllReduceParams {
    size_t    elts_total;
    size_t    elts_per_rank;
    size_t    elts_per_block;
    size_t    rank_offset;
    size_t    rank, local_rank, node_id;
    uint32_t  barrier_flag;
    uint32_t* peer_barrier_ptrs[RANKS_PER_NODE];
    T*        peer_comm_buffer_ptrs[RANKS_PER_NODE];
    T*        local_output_buffer_ptr;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t hadd2(const uint32_t& a, const uint32_t& b)
{
    uint32_t c;
    asm volatile("add.f16x2 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
    return c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t fadd(const uint32_t& a, const uint32_t& b)
{
    uint32_t c;
    asm volatile("add.f32 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
    return c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void ld_flag_acquire(uint32_t& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
template<typename T>
struct ARTypeConverter {
    using Type = uint4;
};

#ifdef ENABLE_BF16
template<>
struct ARTypeConverter<__hip_bfloat16> {
    using Type = bf168;
};
#endif

// add two 128b data
template<typename T_IN, typename T_COMP>
inline __device__ T_IN add128b(T_IN a, T_IN b);

template<>
inline __device__ uint4 add128b<uint4, uint16_t>(uint4 a, uint4 b)
{
    uint4 c;
    c.x = hadd2(a.x, b.x);
    c.y = hadd2(a.y, b.y);
    c.z = hadd2(a.z, b.z);
    c.w = hadd2(a.w, b.w);
    return c;
}

template<>
inline __device__ uint4 add128b<uint4, uint32_t>(uint4 a, uint4 b)
{
    uint4 c;
    c.x = fadd(a.x, b.x);
    c.y = fadd(a.y, b.y);
    c.z = fadd(a.z, b.z);
    c.w = fadd(a.w, b.w);
    return c;
}

#ifdef ENABLE_BF16
template<>
inline __device__ bf168 add128b<bf168, __hip_bfloat16>(bf168 a, bf168 b)
{
    bf168 c;
    c.x = bf16hadd2(a.x, b.x);
    c.y = bf16hadd2(a.y, b.y);
    c.z = bf16hadd2(a.z, b.z);
    c.w = bf16hadd2(a.w, b.w);
    return c;
}
#endif

// init 128bits data with 0
template<typename T>
inline __device__ T init_packed_type();

template<>
inline __device__ uint4 init_packed_type()
{
    return make_uint4(0u, 0u, 0u, 0u);
}

#ifdef ENABLE_BF16
template<>
inline __device__ bf168 init_packed_type()
{
    bf168  val;
    uint4& val_u = reinterpret_cast<uint4&>(val);
    val_u        = make_uint4(0u, 0u, 0u, 0u);
    return val;
}
#endif

template<typename T>
static __global__ void oneShotAllReduceKernel(AllReduceParams<T> params)
{
    // The block index.
    const int bidx = blockIdx.x;
    // The thread index with the block.
    const int tidx = threadIdx.x;

    // The number of elements packed into one for comms
    static constexpr int NUM_ELTS = std::is_same<T, uint32_t>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
    // The end of the segment computed by that block.

    // size_t max_offset = std::min((bidx + 1) * params.elts_per_block, params.elts_per_rank);
    size_t max_offset = min((bidx + 1) * params.elts_per_block, params.elts_per_rank);

    // Synchronize the ranks.
    volatile uint32_t* barrier_d = params.peer_barrier_ptrs[params.local_rank];
    if (tidx < RANKS_PER_NODE) {
        // The 1st block notifies the other ranks.
        if (bidx == 0) {
            params.peer_barrier_ptrs[tidx][params.local_rank] = params.barrier_flag;
        }

        // Busy-wait until all ranks are ready.
        while (barrier_d[tidx] < params.barrier_flag) {}
    }

    // Make sure we can move on...
    __syncthreads();

    // The source pointers. Distributed round-robin for the different warps.
    const T* src_d[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
        int rank  = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii] = params.peer_comm_buffer_ptrs[rank];
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = offset; iter_offset < max_offset; iter_offset += blockDim.x * NUM_ELTS) {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][iter_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the destination buffer.
        reinterpret_cast<PackedType*>(&params.local_output_buffer_ptr[iter_offset])[0] = sums;
    }
}

template<typename T>
static __global__ void twoShotAllReduceKernel(AllReduceParams<T> params)
{

    // The block index.
    const int bidx = blockIdx.x;
    // The thread index with the block.
    const int tidx = threadIdx.x;

    // The number of elements packed into one for comms
    static constexpr int NUM_ELTS = std::is_same<T, uint32_t>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS + params.rank_offset;
    // The end of the segment computed by that block.
    size_t max_offset = min(offset + params.elts_per_block, params.elts_total);

    // Synchronize the ranks.
    volatile uint32_t* barrier_d = params.peer_barrier_ptrs[params.local_rank];
    if (tidx < RANKS_PER_NODE) {
        // The 1st block notifies the other ranks.
        if (bidx == 0) {
            params.peer_barrier_ptrs[tidx][params.local_rank] = params.barrier_flag;
        }

        // Busy-wait until all ranks are ready.
        while (barrier_d[tidx] < params.barrier_flag) {}
    }

    // Make sure we can move on...
    __syncthreads();

    // The source pointers. Distributed round-robin for the different warps.
    T* src_d[RANKS_PER_NODE];
    // The destination ranks for round-robin gathering
    size_t dst_rank[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
        int rank     = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii]    = params.peer_comm_buffer_ptrs[rank];
        dst_rank[ii] = rank;
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = offset; local_offset < max_offset; local_offset += blockDim.x * NUM_ELTS) {

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][local_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the local buffer.
        reinterpret_cast<PackedType*>(&src_d[0][local_offset])[0] = sums;
    }

    // sync threads to make sure all block threads have the sums
    __syncthreads();

    // barreris among the blocks with the same idx (release-acuqire semantics)
    if (tidx < RANKS_PER_NODE) {
        // The all blocks notifies the other ranks.
        uint32_t flag_block_offset = RANKS_PER_NODE + bidx * RANKS_PER_NODE;
        st_flag_release(params.barrier_flag, params.peer_barrier_ptrs[tidx] + flag_block_offset + params.local_rank);

        // Busy-wait until all ranks are ready.
        uint32_t  rank_barrier   = 0;
        uint32_t* peer_barrier_d = params.peer_barrier_ptrs[params.local_rank] + flag_block_offset + tidx;
        do {
            ld_flag_acquire(rank_barrier, peer_barrier_d);
        } while (rank_barrier != params.barrier_flag);
    }

    // sync threads to make sure all other ranks has the final partial results
    __syncthreads();

    // Gather all needed elts from other intra-node ranks
    for (size_t local_offset = offset; local_offset < max_offset; local_offset += blockDim.x * NUM_ELTS) {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
            // use round-robin gathering from other ranks
            int offset_rank = local_offset + (dst_rank[ii] - params.local_rank) * params.elts_per_rank;
            reinterpret_cast<PackedType*>(&params.local_output_buffer_ptr[offset_rank])[0] =
                reinterpret_cast<PackedType*>(&src_d[dst_rank[ii]][offset_rank])[0];
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void kernelLaunchConfig(
    int& blocks_per_grid, int& threads_per_block, size_t elts, int kernel_algo, size_t data_type_bytes)
{
    assert(data_type_bytes == 2 || data_type_bytes == 4);
    // NOTE: need to support FP16 and FP32
    size_t elts_per_thread = 16 / data_type_bytes;
    size_t elts_per_warp   = (16 * WARP_SIZE) / data_type_bytes;
    switch (kernel_algo) {
        case 0: {  // one stage all reduce algo
            assert(elts % elts_per_warp == 0);
            if (elts < (elts_per_thread * DEFAULT_BLOCK_SIZE)) {  // local reduce
                threads_per_block = ((elts + elts_per_warp - 1) / elts_per_warp) * WARP_SIZE;
                blocks_per_grid   = 1;
            }
            else {  // local reduce
                if (elts % (elts_per_thread * threads_per_block) == 0) {
                    blocks_per_grid =
                        (elts + elts_per_thread * threads_per_block - 1) / (elts_per_thread * threads_per_block);
                    // NOTE: need to adjust here
                    // 如果一个block不够，就需要多iter几轮
                    if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS) {
                        int iter_factor = 1;
                        while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor) {
                            iter_factor += 1;
                        }
                        blocks_per_grid /= iter_factor;
                    }
                }
                else {
                    int total_threads = elts / elts_per_thread;
                    blocks_per_grid   = 1;
                    while (total_threads % blocks_per_grid != 0
                           || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE) {
                        blocks_per_grid += 1;
                    }
                    threads_per_block = total_threads / blocks_per_grid;
                }
            }
            break;
        }
        case 1: {  // two stage all reduce algo
            int total_threads = elts / RANKS_PER_NODE / RANKS_PER_NODE;
            assert(elts / RANKS_PER_NODE % RANKS_PER_NODE == 0 && total_threads % WARP_SIZE == 0);

            while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE) {
                blocks_per_grid += 1;
            }

            threads_per_block = total_threads / blocks_per_grid;

            // NOTE: need to adjust here
            if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS) {
                int iter_factor = 1;
                while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor) {
                    iter_factor += 1;
                }
                blocks_per_grid /= iter_factor;
            }
            break;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename T>
void invokeOneOrTwoShotAllReduceKernel(AllReduceParams<T>& param, hipStream_t stream)
{
    size_t elts_total      = param.elts_total;
    int    blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;
    int    kernel_algo = 1;
    if (elts_total <= DEFALUT_ALGO_AR_SIZE_THRESHOLD) {
        kernel_algo = 0;
    }

    kernelLaunchConfig(blocks_per_grid, threads_per_block, elts_total, kernel_algo, sizeof(T));

    if (kernel_algo == 0) {
        param.elts_per_rank  = elts_total;
        param.elts_per_block = param.elts_per_rank / blocks_per_grid;
        oneShotAllReduceKernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
    }
    else {
        param.elts_per_rank  = param.elts_total / RANKS_PER_NODE;
        param.elts_per_block = param.elts_per_rank / blocks_per_grid;
        param.rank_offset    = param.rank * param.elts_per_rank;
        twoShotAllReduceKernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
    }
}

template<typename T>
class CustomAllReduceComm {
public:
    CustomAllReduceComm(size_t rank_size, size_t rank){
        param_.barrier_flag = 0;
        // NOTE: assume All Reduce happens within the node (DGX A100)
        param_.rank       = rank_;
        param_.local_rank = rank_;
        param_.node_id    = 0;
    }
    
    ~CustomAllReduceComm(){
        hipPointerAttribute_t comm_buffer_attributes, barrier_attributes;
        check_cuda_error(hipPointerGetAttributes(&comm_buffer_attributes, param_.peer_comm_buffer_ptrs[rank_]));
        check_cuda_error(hipPointerGetAttributes(&barrier_attributes, param_.peer_barrier_ptrs[rank_]));
        if (comm_buffer_attributes.type == 2) {
            check_cuda_error(hipFree(param_.peer_comm_buffer_ptrs[rank_]));
        }
        if (barrier_attributes.type == 2) {
            check_cuda_error(hipFree(param_.peer_barrier_ptrs[rank_]));
        }
    }

    void customAllReduce(size_t elts, hipStream_t stream){
        param_.elts_total   = elts;
        param_.barrier_flag = FLAG(param_.barrier_flag + 1);

        invokeOneOrTwoShotAllReduceKernel<T>(param_, stream);

        // swap back
        // output_tensor_->at(0).data = (const void*)tmp_tensor_data_;
    }

    void allocateAndExchangePeerAccessPointer(
        std::vector<std::shared_ptr<CustomAllReduceComm>>* custom_all_reduce_comms){
            assert(custom_all_reduce_comms->size() == rank_size_);
        assert(rank_ == 0);
        // Enable Peer to Peer Access
        enableP2P(rank_size_);
        for (size_t i = 0; i < rank_size_; i++) {
            check_cuda_error(hipSetDevice(i));
            check_cuda_error(hipMalloc(&(param_.peer_comm_buffer_ptrs[i]), CUSTOM_AR_SIZE_THRESHOLD));
            check_cuda_error(
                hipMalloc(&(param_.peer_barrier_ptrs[i]), rank_size_ * (MAX_ALL_REDUCE_BLOCKS + 1) * sizeof(uint32_t)));
            check_cuda_error(
                hipMemset(param_.peer_barrier_ptrs[i], 0, rank_size_ * (MAX_ALL_REDUCE_BLOCKS + 1) * sizeof(uint32_t)));
            T*        current_peer_comm_buffer_ptr = param_.peer_comm_buffer_ptrs[i];
            uint32_t* current_peer_barrier_ptr     = param_.peer_barrier_ptrs[i];
            // Assume current comm allocates device memory on all ranks (rank_ == 0)
            for (size_t j = 1; j < rank_size_; j++) {
                static_cast<CustomAllReduceComm<T>*>(custom_all_reduce_comms->at(j).get())
                    ->param_.peer_comm_buffer_ptrs[i] = current_peer_comm_buffer_ptr;
                static_cast<CustomAllReduceComm<T>*>(custom_all_reduce_comms->at(j).get())->param_.peer_barrier_ptrs[i] =
                    current_peer_barrier_ptr;
            }
        }

        // Set default local_output_buffer_ptr to local peer_comm_buffer_ptrs
        for (size_t i = 0; i < rank_size_; i++) {
            static_cast<CustomAllReduceComm<T>*>(custom_all_reduce_comms->at(i).get())->param_.local_output_buffer_ptr =
                static_cast<CustomAllReduceComm<T>*>(custom_all_reduce_comms->at(i).get())->param_.peer_comm_buffer_ptrs[i];
        }
    }

    // bool swapInternalBuffer(std::vector<Tensor>* tensor_buffer, size_t elts){
    //     // Check if all reduce elts meet the requirement of custom kernels
    //     // If meet, then swap the local comm buffer ptr with output tensor data pointer (avoid additional
    //     // memory movement)
    //     if (rank_size_ > 1 && elts * sizeof(T) <= CUSTOM_AR_SIZE_THRESHOLD) {
    //         tmp_tensor_data_               = (T*)(tensor_buffer->at(0).data);
    //         output_tensor_                 = tensor_buffer;
    //         tensor_buffer->at(0).data      = param_.peer_comm_buffer_ptrs[rank_];
    //         param_.local_output_buffer_ptr = tmp_tensor_data_;
    //         return true;
    //     }
    //     return false;
    // }

    bool swapInternalBuffer(T* tensor_buffer, size_t elts){
        // Check if all reduce elts meet the requirement of custom kernels
        // If meet, then swap the local comm buffer ptr with output tensor data pointer (avoid additional
        // memory movement)
        if (rank_size_ > 1 && elts * sizeof(T) <= CUSTOM_AR_SIZE_THRESHOLD) {
            tmp_tensor_data_               = tensor_buffer;
            // output_tensor_                 = tensor_buffer;

            // tensor_buffer->at(0).data      = param_.peer_comm_buffer_ptrs[rank_];
            tensor_buffer      = param_.peer_comm_buffer_ptrs[rank_];
            param_.local_output_buffer_ptr = tmp_tensor_data_;
            return true;
        }
        return false;
    }

    void enableP2P(int ngpus){
        int peer_access_available = 0;
        for (int i = 0; i < ngpus; i++) {
            hipSetDevice(i);
            for (int j = 0; j < ngpus; j++) {
                if (i == j) {
                    continue;
                }
                hipDeviceCanAccessPeer(&peer_access_available, i, j);
                // Custom AR Kernels need DGX A100 NVSWITCH connections
                assert(peer_access_available);
                hipDeviceEnablePeerAccess(j, 0);
            }
        }
    }

private:
    AllReduceParams<T>   param_;
    // std::vector<Tensor>* output_tensor_;
    T*                   tmp_tensor_data_;
    size_t               rank_size_;
    size_t               rank_;
};


template<typename T>
void initCustomAllReduceComm(std::vector<std::shared_ptr<CustomAllReduceComm<T>>>* custom_all_reduce_comms,
                             int                                               enable_custom_all_reduce,
                             size_t                                            rank_size)
{
//     if (custom_all_reduce_comms == 0) {
//         // don't use custom all reduce kernels, fall back to NCCL
//         for (size_t i = 0; i < rank_size; i++) {
//             custom_all_reduce_comms->push_back(nullptr);
//         }
//         return;
//     }

//     if (rank_size != RANKS_PER_NODE) {
// #ifdef BUILD_MULTI_GPU
//         if (rank_size > 1) {
//             FT_LOG_WARNING("Custom All Reduce only supports 8 Ranks currently. Using NCCL as Comm.");
//         }
// #else
//         FT_CHECK_WITH_INFO(rank_size == 1,
//                            fmtstr("Custom All Reduce only supports 8 Ranks currently, got rank_size %ld. FT needs "
//                                   "the NCCL library to communicate among devices but has built without NCCL. "
//                                   "Please use the flag -DBUILD_MULTI_GPU=ON when compiling.",
//                                   rank_size));
// #endif
//         for (size_t i = 0; i < rank_size; i++) {
//             custom_all_reduce_comms->push_back(nullptr);
//         }
//         return;
//     }

// #if defined(CUDART_VERSION) && CUDART_VERSION >= 11020
    for (size_t i = 0; i < rank_size; i++) {
        custom_all_reduce_comms->push_back(std::make_shared<CustomAllReduceComm<T>>(rank_size, i));
    }
    custom_all_reduce_comms->at(0)->allocateAndExchangePeerAccessPointer(custom_all_reduce_comms);
// #else
//     FT_LOG_WARNING("Custom All Reduce is not supported before CUDA 11.2. Using NCCL as Comm.");
//     for (size_t i = 0; i < rank_size; i++) {
//         custom_all_reduce_comms->push_back(nullptr);
//     }
// #endif
}

template<typename T>
void test(){
    int num_gpu = 8; 
    size_t elem_num = 1024; 
    std::vector<hipStream_t> stream_vec{}; 
    for(int i = 0; i < num_gpu; i++){
        hipSetDevice(i); 
        hipStream_t stream; 
        hipStreamCreate(&stream); 
        stream_vec.push_back(stream); 
    }

    std::vector<std::shared_ptr<CustomAllReduceComm<T>>> custom_all_reduce_comms; 

    initCustomAllReduceComm<T>(&custom_all_reduce_comms, true/*enable_custom_all_reduce*/, num_gpu); 

    std::vector<T*> data_vec{}; 
    for(int i = 0; i < num_gpu; i++){
        hipSetDevice(i); 
        T *x; 
        hipMalloc(&x, sizeof(T) * elem_num); 
        data_vec.push_back(x); 
    }

    for(int i = 0; i < num_gpu; i++){
        hipSetDevice(i); 
        custom_all_reduce_comms[i]->swapInternalBuffer(data_vec[i], elem_num); 
    }

    for(int i = 0; i < num_gpu; i++){
        custom_all_reduce_comms[i]->customAllReduce(elem_num, stream_vec[i]); 
    }

    for(int i = 0; i < num_gpu; i++){
        hipSetDevice(i); 
        hipFree(data_vec[i]); 
    }

    for(int i = 0; i < num_gpu; i++){
        hipSetDevice(i); 
        hipStreamDestroy(stream_vec[i]); 
    }

}

int main(){
    test<uint32_t>(); 
}